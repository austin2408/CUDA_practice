#include <stdio.h>
#include <hip/hip_runtime.h>
#include<iostream>

#include <math.h>

#define A_row 5
#define A_col 2
#define B_row 2
#define B_col 4

__global__ void gpu_product(float* A, float* B, float* result){
    // 0~4
    int row = threadIdx.x;

    // 0~3
    int col = threadIdx.y;

    for (int i=0; i<A_col; i++){
        // printf("row : %d || col : %d || i : %d\n",row*B_col + col, A[row*A_col + i], B[i*B_col + col]);

        result[row*B_col + col] += A[row*A_col + i]*B[i*B_col + col];
    }

}

// time function
double ms_time(){
    return (double)clock()/CLOCKS_PER_SEC*1000.0;
}

int main(){

    float A_m[A_row][A_col], B_m[B_row][B_col], C_m[A_row][B_col];

    float* A, * B, * C;

    // set A matrix
    printf("a Matrix: \n");
	for (int i = 0; i < A_row; i++)
	{
		for (int j = 0; j < A_col; j++)
		{
			A_m[i][j] = (float)rand()/RAND_MAX*2-1;
            // A_m[i][j] = i;
			printf("%f   ", (float)A_m[i][j]);
		}
		printf("\n");
	}
    printf("\n");

    // Set B matrix
    printf("b Matrix: \n");
	for (int i = 0; i < B_row; i++)
	{
		for (int j = 0; j < B_col; j++)
		{
			B_m[i][j] = (float)rand()/RAND_MAX*2-1;
            // B_m[i][j] = j;
			printf("%f   ", (float)B_m[i][j]);
		}
		printf("\n");
	}
    printf("\n");

    // Define device's (GPU) memory
    hipMalloc((void **)&A, A_row * A_col * sizeof(float));
    hipMalloc((void **)&B, B_row * B_col * sizeof(float));
    hipMalloc((void **)&C, A_row * B_col * sizeof(float));

    // pass data to device
    hipMemcpy(A, A_m, A_row * A_col * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B, B_m, B_row * B_col * sizeof(float), hipMemcpyHostToDevice);

    dim3 numBlock(A_row, B_col, 1);

    // implement kernel
    double t0 = ms_time();
    gpu_product<<<1, numBlock>>>(A, B, C);
    double device_time = ms_time() - t0;

    hipDeviceSynchronize();

    // pass data to host
    hipMemcpy(C_m, C, A_row * B_col * sizeof(float), hipMemcpyDeviceToHost);

    // Show result
    printf("c Matrix: \n");
	for (int i = 0; i < A_row; i++)
	{
		for (int j = 0; j < B_col; j++)
		{
			printf("%f   ", (float)C_m[i][j]);
		}
		printf("\n");
	}
    printf("Time of gpu : %g\n", device_time);

    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;



}